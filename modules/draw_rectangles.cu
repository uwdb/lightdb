#include "hip/hip_runtime.h"
#include "../core/utility/include/Rectangle.h"

extern "C"
__global__
void draw_rectangles(
        unsigned char* nv12output,
        const unsigned int height,
        const unsigned int width,
        const unsigned int pitch,
        const lightdb::Rectangle *rectangles,
        const unsigned int rectangle_count,
        const unsigned int line_width) {
    const int im_x = blockDim.x * blockIdx.x + threadIdx.x;
    const int im_y = blockDim.y * blockIdx.y + threadIdx.y;
    const unsigned int rectangle_id = blockDim.z * blockIdx.z + threadIdx.z;

    if(rectangle_id < rectangle_count && im_x < width && im_y < height) {
        const lightdb::Rectangle &b = rectangles[rectangle_id];
        const unsigned int output_luma_offset = im_x + im_y * pitch;
        const unsigned int output_luma_size = height * pitch;
        const unsigned int output_chroma_offset = output_luma_size + im_x + (im_y / 2) * pitch;

        const bool on_left_border =   im_x >= b.x &&
                                      im_x < b.x + line_width &&
                                      im_y >= b.y  &&
                                      im_y <= b.y + b.height;
        const bool on_right_border =  im_x > b.x + b.width - line_width &&
                                      im_x <= b.x + b.width &&
                                      im_y >= b.y &&
                                      im_y <= b.y + b.height;

        const bool on_top_border =    im_y >= b.y &&
                                      im_y < b.y + line_width &&
                                      im_x >= b.x + line_width &&
                                      im_x <= b.x + b.width - line_width;
        const bool on_bottom_border = im_y > b.y + b.height - line_width &&
                                      im_y <= b.y + b.height &&
                                      im_x >= b.x + line_width &&
                                      im_x <= b.x + b.width - line_width;

        if (b.width > 0 && (on_top_border || on_bottom_border || on_left_border || on_right_border)) {
            nv12output[output_luma_offset] = 76;
            nv12output[output_chroma_offset] = 64;
        }
    }
}
